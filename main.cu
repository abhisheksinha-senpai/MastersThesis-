#include "hip/hip_runtime.h"
#include "Helper.cuh"

extern unsigned int SCR_WIDTH;
extern unsigned int SCR_HEIGHT;

extern float Ct;
extern float Cl;

extern Vertex **nodeLists;
extern int *vertex_size_per_mesh;
extern float *rho_gpu, *ux_gpu, *uy_gpu, *uz_gpu;
extern float *Fx_gpu, *Fy_gpu, *Fz_gpu;

int main(int argc, char* argv[])
{
    hipDeviceReset();
    hipSetDevice(0);
    getDeviceInfo();

    int NX = 64;
    int NY = 64;
    int NZ = 64;

    float Re_lattice = 10000.0f;
    float viscosity =1.48e-5f;
    float spring_constant = 0.005f;

    float *rho, *ux, *uy, *uz;
    Vertex **nodeData;

    glm::f32vec3 mod_scale = glm::f32vec3(8, 8, 8);
    glm::f32vec3 mod_origin = glm::f32vec3(NX/2, NY/2, NZ/2);

    glm::f32vec3 dis_scale = glm::f32vec3(2.0f/NX, 2.0f/NY, 2.0f/NZ);
    ResourceManager r_manager;
    Shader ourShader;
    Model ourModel;
    GLFWwindow* window;

    display_init(&window);
    model_init(r_manager, ourShader, ourModel, NX, NY, NZ, mod_scale, mod_origin);
    domain_init(NX, NY, NZ, &rho, &ux, &uy, &uz);
    ParticleSystem myfluid(NX, NY, NZ, mod_scale, &rho[0]);
    Geometry fluidDomain = Geometry(1.0f);

    vertex_size_per_mesh = (int *)malloc(ourModel.meshes.size()*sizeof(int));
    nodeLists = (Vertex**)malloc(ourModel.meshes.size()*sizeof(Vertex *));
    nodeData = (Vertex**)malloc(ourModel.meshes.size()*sizeof(Vertex *));

    int num_mesh = ourModel.meshes.size();
    hipStream_t streams[num_mesh];
    for(int i=0;i<num_mesh;i++)
    {
        nodeData[i] = ourModel.meshes[i].vertices.data();
        vertex_size_per_mesh[i] = ourModel.meshes[i].vertices.size();
        hipStreamCreate(&streams[i]);
    }

    float total_size_allocated = 0;
    total_size_allocated += LB_init(NX, NY, NZ, Re_lattice, viscosity, &rho, &ux, &uy, &uz, streams);
    total_size_allocated += IBM_init(NX, NY, NZ, num_mesh, nodeData, streams, spring_constant);
    
    Softbody monkey = Softbody(nodeData[0], vertex_size_per_mesh[0], ourModel.meshes[0].edges.data(), ourModel.meshes[0].edges.size(), 0.0f, 0.0f, Cl);
    checkCudaErrors(hipMemcpy((void *)nodeLists[0], (void *)nodeData[0], vertex_size_per_mesh[0]*sizeof(Vertex), hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());

    float byte_per_GB = powf(1024.0f, 3);
    float Uc = (Re_lattice*viscosity/2.0f);
    printf("Characteristic velocity %f\n", Uc);
    printf("Total memory allocated in GPU: %f GB\n",total_size_allocated/byte_per_GB );
    printf("...............................................................................\n");
    printf("Starting simulation .....\n");
    time_t cur_time1 = clock();
    time_t cur_time2 = clock();
    time_t start_time = clock();
    int KK = 0;
    float time_elapsed = 0.0f;
    while(!glfwWindowShouldClose(window))
    {
        if((((float)(clock() - cur_time2))/CLOCKS_PER_SEC>0.0f) && (((float)(clock() - start_time))/CLOCKS_PER_SEC>2.00f )&& KK++<10000)
        // //if(KK++<2)
        {
            float del_time = Ct;//((clock() - (float)cur_time2)/CLOCKS_PER_SEC);
            LB_simulate_RB(NX, NY, NZ, Ct, IBM_force_spread_RB, IBM_vel_spread_RB, 128, num_mesh, streams);

            checkCudaErrors(hipMemcpy((void *)nodeData[0], (void *)nodeLists[0], vertex_size_per_mesh[0]*sizeof(Vertex), hipMemcpyDeviceToHost));
            checkCudaErrors(hipDeviceSynchronize());

            for(int i=0;i<1;i++)
            {
                monkey.preSolve(del_time/1.0f, glm::f32vec3(NX, NY, NZ), Ct, Cl);
                monkey.SolveEdges(del_time/1.0f);
                monkey.postSolve(del_time/1.0f);
            }

            checkCudaErrors(hipMemcpy((void *)nodeLists[0], (void *)nodeData[0], vertex_size_per_mesh[0]*sizeof(Vertex), hipMemcpyHostToDevice));
            checkCudaErrors(hipDeviceSynchronize());


            cur_time2 = clock();
            time_elapsed += Ct;
            
        }
        if(((float)(clock() - cur_time1))/CLOCKS_PER_SEC>1/30.0f)
        {
            display( rho, ux, uy, uz,
                    rho_gpu, ux_gpu, uy_gpu, uz_gpu,
                    NX, NY, NZ, 
                    myfluid, mod_scale, dis_scale,
                    &window, ourShader, ourModel, fluidDomain);
            cur_time1 = clock();
        }
       
    }
    
    IBM_cleanup(num_mesh);

    LB_cleanup();

    scene_cleanup(nodeLists, nodeData, vertex_size_per_mesh, rho, ux, uy, uz);

    for(int i=0;i<num_mesh; i++)
        hipStreamDestroy(streams[i]);

    return 0;
}