#include "hip/hip_runtime.h"
#include "Helper.cuh"

unsigned int SCR_WIDTH = 800;
unsigned int SCR_HEIGHT = 600;
float deltaTime = 0.0f;
float lastFrame = 0.0f;
float camradius = 7.0f;
float cameraspeed = 0.02f;
float camX = camradius;
float camY = 0.0f;
float camZ = 0.0f;
bool firstMouse = true;
float yaw = -90.0f;
float pitch = 0.0f;
float lastX = SCR_WIDTH / 2.0;
float lastY = SCR_HEIGHT / 2.0;
float fov = 45.0f;

glm::vec3 cameraPos = glm::vec3(camX, camY, camZ);
glm::vec3 cameraFront = glm::vec3(-1.0f, 0.0f, 0.0f);
glm::vec3 cameraUp = glm::vec3(0.0f, 1.0f, 0.0f);

glm::mat4 view = glm::mat4(1.0f);
glm::mat4 model = glm::mat4(1.0f);
glm::mat4 proj = glm::mat4(1.0f);


__host__ void mouse_callback(GLFWwindow* window, double xposIn, double yposIn);
__host__ void framebuffer_size_callback(GLFWwindow* window, int width, int height);

__host__ void display_init(GLFWwindow** window)
{
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    (*window) = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);

    if (*window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return;
    }
    glfwMakeContextCurrent(*window);
    glfwSetFramebufferSizeCallback(*window, framebuffer_size_callback);
    glfwSetCursorPosCallback(*window, mouse_callback);
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return;
    }
    stbi_set_flip_vertically_on_load(true);
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GL_LESS);
    glfwSetInputMode(*window, GLFW_CURSOR, GLFW_CURSOR_NORMAL );
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
    glEnable( GL_BLEND );
    printf("%s %s %s\n", glGetString(GL_VERSION), glGetString(GL_VENDOR), glGetString(GL_RENDERER));
    printf("Display initialized.....\n");
}

__host__ void model_init(ResourceManager &r_manager, Shader &ourShader, Model &ourModel, int NX, int NY, int NZ,
glm::f32vec3 scale, glm::f32vec3 origin)
{
    r_manager.load_shader("resources/shaders/vertex/model_shader.vs", "VERTEX", ourShader.vertex_shader);
    r_manager.load_shader("resources/shaders/fragment/model_shader.fs", "FRAGMENT", ourShader.fragment_shader);
    ourShader.create_vs_shader(ourShader.vertex_shader.c_str());
    ourShader.create_fs_shader(ourShader.fragment_shader.c_str());
    ourShader.compile();
    std::string model_name = "resources/BlenderModels/Cube_8cuts.obj";
    ourModel = Model((char *)model_name.c_str(), scale, origin);
    printf("Model initialized.....\n");
}

void domain_init(int NX, int NY, int NZ,
                float **rho, float **ux, float **uy,float **uz)
{
    int sz = NX*NY*NZ*sizeof(float);
    *rho = (float *)malloc(sz);
    *ux =  (float *)malloc(sz);
    *uy =  (float *)malloc(sz);
    *uz =  (float *)malloc(sz);

    memset(*rho, 0, sz);
    memset(*ux, 0, sz);
    memset(*uy, 0, sz);
    memset(*uz, 0, sz);
    
    int loc = 0, X1, Y1, Z1;
    for(int j=0;j<NY;j++)
    {
        for(int i=0;i<NX;i++)
        {
            for(int k=0;k<NZ;k++)
            {
                loc = i+j*NX+k*NX*NY;
                if(i == 0 || j == 0 || k == 0 || i == NX-1 || j == NY-1 || k == NZ-1)
                {
                    (*rho)[loc] = 99999.0f;
                    (*ux)[loc] = 0.0f;
                    (*uy)[loc] = 0.0f;
                    (*uz)[loc] = 0.0f;
                }
                else
                {
                    // if(j==3*NY/4 && j<8*NY/9 && i<8*NX/9 && i>NX/4 && k<8*NZ/9 && k>NZ/4)
                    // if(j>7*NY/9 && j<8*NY/9)
                    if(j<1*NY/20)
                        (*rho)[loc] = 1.0f;
                    // if(powf((i-NX/2), 2.0f)+powf((j-5*NY/8), 2.0f)+powf((k-NZ/2), 2.0f)<powf(NX/16, 2.0f))
                    //     (*rho)[loc] = 1.0f;
                    // else if((j<NY*1/4))// && (i>NX/4 && i<3*NX/4) && (k<3*NZ/4 && k>NZ/4))
                    //     (*rho)[loc] = 1.0f;
                    else
                        (*rho)[loc] = 0.001f;
                    (*ux)[loc] = 0.0f;
                    (*uy)[loc] = 0.0f;
                    (*uz)[loc] = 0.0f;
                }
            }
        }
    }
    printf("Domain initialized...\n");
}

__host__ void scene_init(float *rho_gpu, float *ux_gpu, float *uy_gpu, float *uz_gpu,
                         float *rho, float *ux, float *uy, float *uz, 
                         int NX, int NY, int NZ)
{
    int sz = NX*NY*NZ*sizeof(float);
    checkCudaErrors(hipMemcpy(temp_cell_type_gpu, rho, sz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ux_gpu, ux, sz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(uy_gpu, uy, sz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(uz_gpu, uz, sz, hipMemcpyHostToDevice));
    printf("Scene initialized.....\n");
}

__host__ void scene_cleanup(Vertex **nodeLists, Vertex **nodeData, int *vertex_size_per_mesh,
                            float *rho, float *ux, float *uy, float *uz)
{
    free(rho);
    free(ux);
    free(uy);
    free(uz);
    free(nodeLists);
    free(nodeData);
    free(vertex_size_per_mesh);
}

__host__ void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

__host__ void processInput(GLFWwindow* window)
{
    float currentFrame = glfwGetTime();
    deltaTime = currentFrame - lastFrame;
    lastFrame = currentFrame;

    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);

    const float cameraSpeed = 5.0f * deltaTime; // adjust accordingly
    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
        cameraPos += cameraSpeed * cameraFront;
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
        cameraPos -= cameraSpeed * cameraFront;
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
        cameraPos -= glm::normalize(glm::cross(cameraFront, cameraUp)) * cameraSpeed;
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
        cameraPos += glm::normalize(glm::cross(cameraFront, cameraUp)) * cameraSpeed;
}

__host__ void mouse_callback(GLFWwindow* window, double xposIn, double yposIn)
{
    if (glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_RIGHT) == GLFW_PRESS)
    {
        float xpos = static_cast<float>(xposIn);
        float ypos = static_cast<float>(yposIn);

        if (firstMouse)
        {
            lastX = xpos;
            lastY = ypos;
            firstMouse = false;
        }

        float xoffset = xpos - lastX;
        float yoffset = lastY - ypos; // reversed since y-coordinates go from bottom to top
        lastX = xpos;
        lastY = ypos;

        float sensitivity = 0.25f; // change this value to your liking
        xoffset *= sensitivity;
        yoffset *= sensitivity;

        yaw += xoffset;
        pitch += yoffset;

        // make sure that when pitch is out of bounds, screen doesn't get flipped
        if (pitch > 89.0f)
            pitch = 89.0f;
        if (pitch < -89.0f)
            pitch = -89.0f;

        glm::vec3 front;
        front.x = cos(glm::radians(yaw)) * cos(glm::radians(pitch));
        front.y = sin(glm::radians(pitch));
        front.z = sin(glm::radians(yaw)) * cos(glm::radians(pitch));

        cameraFront = glm::normalize(front);
    }
    else if (glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_RIGHT) == GLFW_RELEASE)
    {
        lastX = SCR_WIDTH / 2.0;
        lastY = SCR_HEIGHT / 2.0;
        firstMouse = true;
    }
}

__host__ void draw_model( GLFWwindow* window, Shader& shader, Model& objmodel, glm::f32vec3 scale)
{
    shader.use();
    // view/projection transformations
    view = glm::lookAt(cameraPos, cameraPos+cameraFront, cameraUp);
    proj = glm::perspective(glm::radians(45.0f), (float)SCR_WIDTH / (float)SCR_HEIGHT, 0.1f, 1000.0f);

    glUniformMatrix4fv(glGetUniformLocation(shader.get_shader_pgm(), "view"), 1, GL_FALSE, glm::value_ptr(view));
    glUniformMatrix4fv(glGetUniformLocation(shader.get_shader_pgm(), "projection"), 1, GL_FALSE, glm::value_ptr(proj));
    // render the loaded model
    objmodel.Draw(shader, scale);
    model = glm::mat4(1);
}

__host__ void transfer_fluid_data(float *rho, float*ux, float *uy,float *uz,
                                  float *rho_gpu, float *ux_gpu, float*uy_gpu, float* uz_gpu, 
                                  int NX, int NY, int NZ)
{
    int sz = NX*NY*NZ*sizeof(float);
    hipMemcpy(rho, (void *)mass_gpu, sz, hipMemcpyDeviceToHost);
    hipMemcpy(ux, (void *)Fx_gpu, sz, hipMemcpyDeviceToHost);
    hipMemcpy(uy, (void *)Fy_gpu, sz, hipMemcpyDeviceToHost);
    hipMemcpy(uz, (void *)Fz_gpu, sz, hipMemcpyDeviceToHost);
}

__host__ void draw_fluid(float *rho, float*ux, float *uy, float *uz,
                         float *rho_gpu, float *ux_gpu, float*uy_gpu, float* uz_gpu,
                         int NX, int NY, int NZ, 
                         ParticleSystem &fluid, glm::f32vec3 model_scale, glm::f32vec3 dis_scale)
{
    transfer_fluid_data(rho, ux, uy, uz,
                        mass_gpu, ux_gpu, uy_gpu, uz_gpu,
                        NX, NY, NZ);
    
    fluid.update_particles(NX, NY, NZ, rho, ux, uy, uz, model_scale);
    fluid.draw_particles(SCR_WIDTH, SCR_HEIGHT, cameraPos, cameraFront, cameraUp, dis_scale);
}
int n = 0;
__host__ void display ( float *rho, float*ux, float *uy, float *uz,
                        float *rho_gpu, float *ux_gpu, float*uy_gpu, float* uz_gpu,
                        int NX, int NY, int NZ, 
                        ParticleSystem &fluid, glm::f32vec3 mod_scale, glm::f32vec3 dis_scale,
                        GLFWwindow** window, Shader& shader, Model &model, Geometry &fluidDomain)
{
    glClearColor(0.35f, 0.15f, 0.35f, 0.05f);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    draw_model( *window, shader, model, dis_scale);

    fluidDomain.draw_geometry(SCR_WIDTH, SCR_HEIGHT, cameraPos, cameraFront, cameraUp);
    
    draw_fluid(rho, ux, uy,uz, mass_gpu, ux_gpu, uy_gpu, uz_gpu, NX, NY, NZ, fluid, mod_scale, dis_scale);

    processInput(*window);
    glfwPollEvents();
    glfwSwapBuffers(*window);
}